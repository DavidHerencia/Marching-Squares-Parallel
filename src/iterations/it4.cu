#include "hip/hip_runtime.h"
#include "../marching_squares.hpp"
#include <stdio.h>
#include <vector>
#include <iostream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

struct CudaLineSegment
{
    double x1, y1, x2, y2; // Coordenadas de los extremos
    int id;

    __host__ __device__ CudaLineSegment() : x1(0.0), y1(0.0), x2(0.0), y2(0.0), id(-1) {}
    __host__ __device__ CudaLineSegment(double a, double b, double c, double d, int segment_id = -1)
        : x1(a), y1(b), x2(c), y2(d), id(segment_id) {}
};

struct CellOutput
{
    CudaLineSegment lines[2]; // Máximo 2 líneas por celda en marching squares

    int line_count;
    __host__ __device__ CellOutput() : line_count(0) {}
};

// Funciones device para Marching Squares
__device__ double cudaFromIndexSpace(int index, double min_v, double max_v, int grid_size)
{
    return min_v + (max_v - min_v) * double(index) / double(grid_size);
}

__device__ double cudaInterp(double A, double B)
{
    double diff = A - B;
    if (fabs(diff) < 1e-12)
        return 0.5; // Evitar división por cero
    return A / diff;
}

// Device versions of mathematical functions
__device__ double evaluateFunction(FunctionID funcId, double x, double y)
{
    const double PI = 3.14159265358979323846;
    
    switch (funcId) {
        case CIRCLE_FUNC:
            return x * x + y * y - 1.5;
            
        case CHECKERBOARD_FUNC:
            return sin(2 * PI * x) * sin(2 * PI * y);
            
        case TANGENT_FUNC:
            return tan(((x * x)) + (y * y));
            
        case NOISY_FUNC: {
            double fx = cos(2 * x) * cos(PI * x);
            double fy = sin(y) * sin(PI * y);
            return fx + fy;
        }
        
        case WAVE_FUNC:
            return sin(5 * x) + cos(5 * y) - 0.5;
            
        case SPIRAL_FUNC: {
            double r = sqrt(x * x + y * y);
            double theta = atan2(y, x);
            return sin(4 * theta + 2 * r) - 0.3;
        }
        
        case MANDELBROT_FUNC:
            return x * x * x - 3 * x * y * y + y * y - 1;
            
        case CAVE_NOISE_FUNC:
            // Simplified noise function - for full SimplexNoise we'd need to port the entire implementation
            // This is a basic approximation
            return sin(x * 12.9898 + y * 78.233) * 43758.5453;
            
        default:
            return x * x + y * y - 1.0; // Default to circle
    }
}

__global__ void cudaMarchingSquares(int grid_size, double min_v, double max_v, FunctionID funcId, CellOutput *d_cell_output)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // fila
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columna

    // VERIFICACIÓN CRÍTICA: Evitar acceso fuera de límites
    if (i >= grid_size || j >= grid_size)
        return;

    int cellIdx = i * grid_size + j;

    // VERIFICACIÓN ADICIONAL: Verificar bounds del array
    if (cellIdx >= grid_size * grid_size)
        return;

    const double DT = (fabs(min_v - max_v)) / double(grid_size);

    CellOutput currentCell;
    currentCell.line_count = 0;

    // Calculate the values at the corners of the grid cell
    double x_sw = cudaFromIndexSpace(j, min_v, max_v, grid_size);
    double x_ne = cudaFromIndexSpace(j + 1, min_v, max_v, grid_size);
    double y_sw = cudaFromIndexSpace(i, min_v, max_v, grid_size);
    double y_ne = cudaFromIndexSpace(i + 1, min_v, max_v, grid_size);

    double A = evaluateFunction(funcId, x_sw, y_sw); // 0 0
    double B = evaluateFunction(funcId, x_ne, y_sw); // 1 0
    double C = evaluateFunction(funcId, x_ne, y_ne); // 1 1
    double D = evaluateFunction(funcId, x_sw, y_ne); // 0 1

    double d_bottom = cudaInterp(A, B) * DT;
    double d_right = cudaInterp(B, C) * DT;
    double d_top = cudaInterp(C, D) * DT;
    double d_left = cudaInterp(D, A) * DT;

    // 4 NEIGHBOURS
    int c_case = ((D > 0) << 3) | ((C > 0) << 2) | ((B > 0) << 1) | (A > 0);

    // Lambda function to add line
    auto addLine = [&currentCell](double x1, double y1, double x2, double y2)
    {
        if (currentCell.line_count < 2)
        {
            currentCell.lines[currentCell.line_count] = CudaLineSegment(x1, y1, x2, y2, currentCell.line_count);
            currentCell.line_count++;
        }
    };

    // Now do cases
    switch (c_case)
    {
    case 0:
    case 15:
        break;
    case 1:
    case 14:
        addLine(x_sw, y_ne - d_left, x_sw + d_bottom, y_sw);
        break;
    case 2:
    case 13:
        addLine(x_sw + d_bottom, y_sw, x_ne, y_sw + d_right);
        break;
    case 3:
    case 12:
        addLine(x_sw, y_ne - d_left, x_ne, y_sw + d_right);
        break;
    case 4:
    case 11:
        addLine(x_ne - d_top, y_ne, x_ne, y_sw + d_right);
        break;
    case 5:
        addLine(x_sw, y_ne - d_left, x_ne - d_top, y_ne);
        addLine(x_sw + d_bottom, y_sw, x_ne, y_sw + d_right);
        break;
    case 6:
    case 9:
        addLine(x_sw + d_bottom, y_sw, x_ne - d_top, y_ne);
        break;
    case 7:
    case 8:
        addLine(x_sw, y_ne - d_left, x_ne - d_top, y_ne);
        break;
    case 10:
        addLine(x_sw + d_bottom, y_ne, x_ne, y_sw + d_right);
        addLine(x_sw, y_ne - d_left, x_sw + d_bottom, y_sw);
        break;
    }

    d_cell_output[cellIdx] = currentCell;
}


vector<LineSegment> marching_squares(FunctionID f, int grid_size, double min_v, double max_v)
{
    // Determine which function we're using
    int numCells = grid_size * grid_size;
    size_t cellOutputSize = numCells * sizeof(CellOutput);
    CellOutput *d_cell_output;

    // VERIFICACIÓN CRÍTICA: Comprobar malloc
    hipError_t mallocError = hipMalloc((void **)&d_cell_output, cellOutputSize);
    if (mallocError != hipSuccess)
    {
        printf("ERROR CRÍTICO: hipMalloc falló: %s\n", hipGetErrorString(mallocError));
        return std::vector<LineSegment>(); // Retornar vector vacío en caso de error
    }

    // Configuración de bloques y threads
    int threadsPerBlockX = 16;
    int threadsPerBlockY = 16;
    dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 dimGrid(
        (grid_size + threadsPerBlockX - 1) / threadsPerBlockX,
        (grid_size + threadsPerBlockY - 1) / threadsPerBlockY);

    // Ejecutar kernel de marching squares
    cudaMarchingSquares<<<dimGrid, dimBlock>>>(grid_size, min_v, max_v, f, d_cell_output);

    // VERIFICACIÓN CRÍTICA: Comprobar errores del kernel
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess)
    {
        printf("ERROR CRÍTICO: Kernel falló: %s\n", hipGetErrorString(kernelError));
        hipFree(d_cell_output);
        return std::vector<LineSegment>();
    }

    hipDeviceSynchronize();

    // VERIFICACIÓN CRÍTICA: Comprobar sincronización
    hipError_t syncError = hipGetLastError();
    if (syncError != hipSuccess)
    {
        printf("ERROR CRÍTICO: Sincronización falló: %s\n", hipGetErrorString(syncError));
        hipFree(d_cell_output);
        return std::vector<LineSegment>();
    }

    std::vector<CellOutput> h_cell_output(numCells);

    // VERIFICACIÓN CRÍTICA: Comprobar memcpy
    hipError_t copyError = hipMemcpy(h_cell_output.data(), d_cell_output, cellOutputSize, hipMemcpyDeviceToHost);
    if (copyError != hipSuccess)
    {
        printf("ERROR CRÍTICO: hipMemcpy falló: %s\n", hipGetErrorString(copyError));
        hipFree(d_cell_output);
        return std::vector<LineSegment>();
    }

    hipFree(d_cell_output);

    std::vector<LineSegment> result;
    for (int i = 0; i < numCells; ++i)
    {
        for (int j = 0; j < h_cell_output[i].line_count; ++j)
        {
            const CudaLineSegment &seg = h_cell_output[i].lines[j];
            result.push_back({seg.x1, seg.y1, seg.x2, seg.y2});
        }
    }
    return result;
}

/*
int main() {
    int grid_size = 512;  // Tamaño del grid para marching squares
    double min_v = -2.0;
    double max_v = 2.0;
    printf("Ejecutando Marching Squares en CUDA con grid %d x %d...\n", grid_size, grid_size);
    // Verificar errores de CUDA
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error CUDA: %s\n", hipGetErrorString(error));
        return -1;
    }
    // Copiar resultados de vuelta al host
    // Contar líneas generadas
    int totalLines = 0;
    int cellsWithLines = 0;
    for (int i = 0; i < numCells; ++i) {
        if (h_cell_output[i].line_count > 0) {
            cellsWithLines++;
            totalLines += h_cell_output[i].line_count;
        }
    }
    printf("\nResultados:\n");
    printf("Total de celdas con líneas: %d\n", cellsWithLines);
    printf("Total de líneas generadas: %d\n", totalLines);
    // Mostrar algunas líneas de ejemplo (primeras 10)
    printf("\nPrimeras 10 líneas generadas:\n");
    int linesShown = 0;
    for (int i = 0; i < numCells && linesShown < 10; ++i) {
        if (h_cell_output[i].line_count > 0) {
            int row = i / grid_size;
            int col = i % grid_size;
            for (int j = 0; j < h_cell_output[i].line_count && linesShown < 10; ++j) {
                LineSegment line = h_cell_output[i].lines[j];
                printf("Celda (%d,%d) - Línea %d: (%.3f,%.3f) -> (%.3f,%.3f)\n",
                       row, col, j, line.p1.x, line.p1.y, line.p2.x, line.p2.y);
                linesShown++;
            }
        }
    }
    // Renderizar a imagen
    printf("\nGenerando imagen de resultado...\n");
    render_cuda_results(h_cell_output, grid_size, "cuda_marching_squares.png", min_v, max_v, 1024, 1024);
    printf("Imagen guardada como: cuda_marching_squares.png\n");
    return 0;
}
*/