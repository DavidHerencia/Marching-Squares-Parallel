#include "hip/hip_runtime.h"
#include "../marching_squares.hpp"
#include "../simplex_noise.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <iostream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


// Funciones device para Marching Squares
__device__ float cudaFromIndexSpace(int index, float min_v, float max_v, int grid_size)
{
    return min_v + (max_v - min_v) * float(index) / float(grid_size);
}

__device__ float cudaInterp(float A, float B)
{
    float diff = A - B;
    if (fabs(diff) < 1e-12)
        return 0.5; // Evitar división por cero
    return A / diff;
}

// Device versions of mathematical functions
__device__ float evaluateFunction(FunctionID funcId, float x, float y)
{
    const float PI = 3.14159265358979323846;
    
    switch (funcId) {
        case CIRCLE_FUNC:
            return x * x + y * y - 1.5;
            
        case CHECKERBOARD_FUNC:
            return sin(2 * PI * x) * sin(2 * PI * y);
            
        case TANGENT_FUNC:
            return tan(((x * x)) + (y * y));
            
        case NOISY_FUNC: {
            float fx = cos(2 * x) * cos(PI * x);
            float fy = sin(y) * sin(PI * y);
            return fx + fy;
        }
        
        case WAVE_FUNC:
            return sin(5 * x) + cos(5 * y) - 0.5;
            
        case SPIRAL_FUNC: {
            float r = sqrt(x * x + y * y);
            float theta = atan2(y, x);
            return sin(4 * theta + 2 * r) - 0.3;
        }
        
        case MANDELBROT_FUNC:
            return x * x * x - 3 * x * y * y + y * y - 1;
            
        case CAVE_NOISE_FUNC: {
            float3 p = make_float3(x, y, 0.0f);
            float noise = cudaNoise::simplexNoise(p, 0.5f, 42);
            return noise * 2.0f - 1.0f; // Normalizar a rango [-1, 1]
        }
            
        default:
            return x * x + y * y - 1.0; // Default to circle
    }
}

__global__ void cudaMarchingSquares(int grid_size, float min_v, float max_v, FunctionID funcId, CellOutput *d_cell_output)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // fila
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columna

    // VERIFICACIÓN CRÍTICA: Evitar acceso fuera de límites
    if (i >= grid_size || j >= grid_size)
        return;

    int cellIdx = i * grid_size + j;

    // VERIFICACIÓN ADICIONAL: Verificar bounds del array
    if (cellIdx >= grid_size * grid_size)
        return;

    const float DT = (fabs(min_v - max_v)) / float(grid_size);

    CellOutput currentCell;
    currentCell.line_count = 0;

    // Calculate the values at the corners of the grid cell
    float x_sw = cudaFromIndexSpace(j, min_v, max_v, grid_size);
    float x_ne = cudaFromIndexSpace(j + 1, min_v, max_v, grid_size);
    float y_sw = cudaFromIndexSpace(i, min_v, max_v, grid_size);
    float y_ne = cudaFromIndexSpace(i + 1, min_v, max_v, grid_size);

    float A = evaluateFunction(funcId, x_sw, y_sw); // 0 0
    float B = evaluateFunction(funcId, x_ne, y_sw); // 1 0
    float C = evaluateFunction(funcId, x_ne, y_ne); // 1 1
    float D = evaluateFunction(funcId, x_sw, y_ne); // 0 1

    float d_bottom = cudaInterp(A, B) * DT;
    float d_right = cudaInterp(B, C) * DT;
    float d_top = cudaInterp(C, D) * DT;
    float d_left = cudaInterp(D, A) * DT;

    // 4 NEIGHBOURS
    int c_case = ((D > 0) << 3) | ((C > 0) << 2) | ((B > 0) << 1) | (A > 0);

    // Lambda function to add line
    auto addLine = [&currentCell](float x1, float y1, float x2, float y2)
    {
        if (currentCell.line_count < 2)
        {
            currentCell.lines[currentCell.line_count] = CudaLineSegment(x1, y1, x2, y2, currentCell.line_count);
            currentCell.line_count++;
        }
    };

    // Now do cases
    switch (c_case)
    {
    case 0:
    case 15:
        break;
    case 1:
    case 14:
        addLine(x_sw, y_ne - d_left, x_sw + d_bottom, y_sw);
        break;
    case 2:
    case 13:
        addLine(x_sw + d_bottom, y_sw, x_ne, y_sw + d_right);
        break;
    case 3:
    case 12:
        addLine(x_sw, y_ne - d_left, x_ne, y_sw + d_right);
        break;
    case 4:
    case 11:
        addLine(x_ne - d_top, y_ne, x_ne, y_sw + d_right);
        break;
    case 5:
        addLine(x_sw, y_ne - d_left, x_ne - d_top, y_ne);
        addLine(x_sw + d_bottom, y_sw, x_ne, y_sw + d_right);
        break;
    case 6:
    case 9:
        addLine(x_sw + d_bottom, y_sw, x_ne - d_top, y_ne);
        break;
    case 7:
    case 8:
        addLine(x_sw, y_ne - d_left, x_ne - d_top, y_ne);
        break;
    case 10:
        addLine(x_sw + d_bottom, y_ne, x_ne, y_sw + d_right);
        addLine(x_sw, y_ne - d_left, x_sw + d_bottom, y_sw);
        break;
    }

    d_cell_output[cellIdx] = currentCell;
}


vector<CellOutput> marching_squares(FunctionID f, int grid_size, double min_v, double max_v)
{
    // Determine which function we're using
    int numCells = grid_size * grid_size;
    size_t cellOutputSize = numCells * sizeof(CellOutput);
    CellOutput *d_cell_output;

    // VERIFICACIÓN CRÍTICA: Comprobar malloc
    hipError_t mallocError = hipMalloc((void **)&d_cell_output, cellOutputSize);
    if (mallocError != hipSuccess)
    {
        printf("ERROR CRÍTICO: hipMalloc falló: %s\n", hipGetErrorString(mallocError));
        return std::vector<CellOutput>(); // Retornar vector vacío en caso de error
    }

    // Configuración de bloques y threads
    int threadsPerBlockX = 16;
    int threadsPerBlockY = 16;
    dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 dimGrid(
        (grid_size + threadsPerBlockX - 1) / threadsPerBlockX,
        (grid_size + threadsPerBlockY - 1) / threadsPerBlockY);

    // Ejecutar kernel de marching squares
    cudaMarchingSquares<<<dimGrid, dimBlock>>>(grid_size, min_v, max_v, f, d_cell_output);

    // VERIFICACIÓN CRÍTICA: Comprobar errores del kernel
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess)
    {
        printf("ERROR CRÍTICO: Kernel falló: %s\n", hipGetErrorString(kernelError));
        hipFree(d_cell_output);
        return std::vector<CellOutput>();
    }

    hipDeviceSynchronize();

    // VERIFICACIÓN CRÍTICA: Comprobar sincronización
    hipError_t syncError = hipGetLastError();
    if (syncError != hipSuccess)
    {
        printf("ERROR CRÍTICO: Sincronización falló: %s\n", hipGetErrorString(syncError));
        hipFree(d_cell_output);
        return std::vector<CellOutput>();
    }

    std::vector<CellOutput> h_cell_output(numCells);

    // VERIFICACIÓN CRÍTICA: Comprobar memcpy
    hipError_t copyError = hipMemcpy(h_cell_output.data(), d_cell_output, cellOutputSize, hipMemcpyDeviceToHost);
    if (copyError != hipSuccess)
    {
        printf("ERROR CRÍTICO: hipMemcpy falló: %s\n", hipGetErrorString(copyError));
        hipFree(d_cell_output);
        return std::vector<CellOutput>();
    }
    hipFree(d_cell_output);
    return h_cell_output;
}

/*
int main() {
    int grid_size = 512;  // Tamaño del grid para marching squares
    float min_v = -2.0;
    float max_v = 2.0;
    printf("Ejecutando Marching Squares en CUDA con grid %d x %d...\n", grid_size, grid_size);
    // Verificar errores de CUDA
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error CUDA: %s\n", hipGetErrorString(error));
        return -1;
    }
    // Copiar resultados de vuelta al host
    // Contar líneas generadas
    int totalLines = 0;
    int cellsWithLines = 0;
    for (int i = 0; i < numCells; ++i) {
        if (h_cell_output[i].line_count > 0) {
            cellsWithLines++;
            totalLines += h_cell_output[i].line_count;
        }
    }
    printf("\nResultados:\n");
    printf("Total de celdas con líneas: %d\n", cellsWithLines);
    printf("Total de líneas generadas: %d\n", totalLines);
    // Mostrar algunas líneas de ejemplo (primeras 10)
    printf("\nPrimeras 10 líneas generadas:\n");
    int linesShown = 0;
    for (int i = 0; i < numCells && linesShown < 10; ++i) {
        if (h_cell_output[i].line_count > 0) {
            int row = i / grid_size;
            int col = i % grid_size;
            for (int j = 0; j < h_cell_output[i].line_count && linesShown < 10; ++j) {
                LineSegment line = h_cell_output[i].lines[j];
                printf("Celda (%d,%d) - Línea %d: (%.3f,%.3f) -> (%.3f,%.3f)\n",
                       row, col, j, line.p1.x, line.p1.y, line.p2.x, line.p2.y);
                linesShown++;
            }
        }
    }
    // Renderizar a imagen
    printf("\nGenerando imagen de resultado...\n");
    render_cuda_results(h_cell_output, grid_size, "cuda_marching_squares.png", min_v, max_v, 1024, 1024);
    printf("Imagen guardada como: cuda_marching_squares.png\n");
    return 0;
}
*/